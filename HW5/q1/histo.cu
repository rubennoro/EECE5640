
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <iostream>

// Set the dataset size
#define N (1 << 12)
#define NUM_BINS 256

double CLOCK(){
	struct timespec t;
	clock_gettime(CLOCK_MONOTONIC, &t);
	return (t.tv_sec * 1000) + (t.tv_nsec*1e-6);
}

// CUDA Kernel
__global__ void histogram(int* histogram, int* data, int n){

	// Determine index, ensure it's less than n
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
	if (idx < n) {
        
		// Get value from data and put it in a bin
		int value = data[idx];
        
		int bin = (value - 1) / (100000 / NUM_BINS);
        
		if (bin >= NUM_BINS) bin = NUM_BINS - 1;
		
		//Atomically add it to main histogram
        atomicAdd(&histogram[bin], 1);  // Atomic operation for thread safety
    }
}

// Random number generation for dataset
void generateData(int* arr){
	for(int i = 0; i < N; i++){
		arr[i] = rand() % 100000 + 1; 
	}
}

int main(int argc, char* argv[]){

	srand(time(NULL));

	// Initialize the random numbers into an array
	int* hostData = new int[N];
	generateData(hostData);

	// Initailize histogram values to 0
	int* hostHist = new int[NUM_BINS]();

	int maxVal = 100000;
	int binWidth = (maxVal) / NUM_BINS;

	// CUDA device variables
	int* deviceData;
	int* deviceHist;

	double start, finish, total;

	// Include the device to host communication in timing
	hipMalloc(&deviceData, N * sizeof(int));
	hipMalloc(&deviceHist, NUM_BINS * sizeof(int));

	start = CLOCK();
	// Copy arrays over to GPU memory
	hipMemcpy(deviceHist, hostHist, NUM_BINS * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceData, hostData, N * sizeof(int), hipMemcpyHostToDevice);

	// Set sizes of parallelization
	int blockSize = 256;  // Optimal for modern GPUs
	int gridSize = (N + blockSize - 1) / blockSize;

	// Execute the kernel
	histogram<<<gridSize, blockSize>>>(deviceHist, deviceData, N);
	hipDeviceSynchronize();
	
	// Include the host to device communication in timing
	hipMemcpy(hostHist, deviceHist, NUM_BINS * sizeof(int), hipMemcpyDeviceToHost);
	
	finish = CLOCK();
	total = finish - start;

	//cudaMemcpy(hostHist, deviceHist, NUM_BINS * sizeof(int), cudaMemcpyDeviceToHost);

	std::cout << "Results: " << std::endl;

	// Print out one element from each class in class ascending order
	for(int i = 0; i < NUM_BINS; i++){
		int lower = 1 + i * binWidth;
		int upper = (i == NUM_BINS - 1) ? maxVal : (lower + binWidth - 1);
		//std::cout << "Class " << i+1 << " (" << lower << " - " << upper << " ) " << hostHist[i] << " elements." << std::endl;
	}
	std::cout << "Time: " << total << std::endl;


	delete[] hostHist;
	delete[] hostData;
    hipFree(deviceData);
    hipFree(deviceHist);
	
	return 0;
}
