#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <iostream>
#include <hip/hip_runtime.h>

// Setting n and the tile size for tiling
#define n 512
#define tileSize 8

double CLOCK(){
	struct timespec t;
	clock_gettime(CLOCK_MONOTONIC, &t);
	return (t.tv_sec * 1000) + (t.tv_nsec*1e-6);
}

__global__ void nn(float (*a)[n][n], float (*b)[n][n]){

    // Shared memory of halo array, covering surrounding indices of each thread
    __shared__ float sharedB[tileSize + 2][tileSize + 2][tileSize + 2];

    // Thread Indices
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;

    // Block Indices
    int bx = 1 + blockIdx.x * tileSize;
    int by = 1 + blockIdx.y * tileSize;
    int bz = 1 + blockIdx.z * tileSize;

    // Calcluating the indices themselves
    int x = bx + tx - 1;
    int y = by + ty - 1;
    int z = bz + tz - 1;

    //Adding values to shared memory halo
    if (x >= 0 && x < n && y >= 0 && y < n && z >= 0 && z < n) {
        // Load data (including halo)
        sharedB[tx][ty][tz] = b[x][y][z];
    }

    // Ensure all threads have reached this point before continuation
    __syncthreads();

    // The edges will NOT be modified
    if (tx > 0 && tx <= tileSize && 
        ty > 0 && ty <= tileSize && 
        tz > 0 && tz <= tileSize) {
        
        // Convert back to global index within 3D array
        int gi = bx + tx - 1; 
        int gj = by + ty - 1;
        int gk = bz + tz - 1;

        if (gi >= 1 && gi < n-1 && gj >= 1 && gj < n-1 && gk >= 1 && gk < n-1) {
            a[gi][gj][gk] = 0.75f * (
                sharedB[tx-1][ty][tz] + sharedB[tx+1][ty][tz] +  // b[x±1][y][z]
                sharedB[tx][ty-1][tz] + sharedB[tx][ty+1][tz] +  // b[x][y±1][z]
                sharedB[tx][ty][tz-1] + sharedB[tx][ty][tz+1]    // b[x][y][z±1]
            );
        }
    }

    // Ensure all threads have reached this point before continuation
    __syncthreads();
    

}

int main(int argc, char* argv[]){

    double start, end, total;

    //dim3 dimensions = dim3(n, n, n);
    float a[n][n][n], b[n][n][n];
    for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++){
            for(int k = 0; k < n; k++){
                b[i][j][k] = 2.0f;
                a[i][j][k] = 0.0f;
            }
        }
    }

    // Allocate device memory for 3D arrays
    float (*cuA)[n][n], (*cuB)[n][n];
    hipMalloc(&cuA, sizeof(float)*n*n*n);
    hipMalloc(&cuB, sizeof(float)*n*n*n);
    
    // Copy data to device
    hipMemcpy(cuB, b, sizeof(float)*n*n*n, hipMemcpyHostToDevice);
    hipMemcpy(cuA, a, sizeof(float)*n*n*n, hipMemcpyHostToDevice);

    // Dimensions
    /*
    For 32 x 32 x 32:
    - 6 x 6 x 6 threads per block
    - 8 x 8 x 8 blocks in grid
    */
    dim3 gridDim((n -2 + tileSize - 1) / tileSize, (n - 2+ tileSize - 1) / tileSize, (n - 2+ tileSize - 1) / tileSize);
    dim3 blockDim(tileSize + 2, tileSize + 2, tileSize + 2);

    // track start of operation
    start = CLOCK();

    // Execute kernel
    for(int i = 0; i < 1000; i++){
        nn<<<gridDim, blockDim>>>(cuA, cuB);
    }
    hipDeviceSynchronize();

    // track end of operation
    end = CLOCK();

    // copy memory back to device 3d array
    hipMemcpy(a, cuA, sizeof(float) * n * n * n, hipMemcpyDeviceToHost);

    // Time calculations and printing
    total = end - start;
    std::cout << "Time: " << total << " ms" << std::endl;
    
    // Checking answers
    for(int i = 0; i < 10; i++){
        //std::cout << a[1][1][i] << std::endl;
    }

    hipFree(cuA);
    hipFree(cuB);

    return 0;
}