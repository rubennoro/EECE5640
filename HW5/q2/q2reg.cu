#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <iostream>
#include <hip/hip_runtime.h>

// Setting n and the tile size for tiling
#define n 512

double CLOCK(){
	struct timespec t;
	clock_gettime(CLOCK_MONOTONIC, &t);
	return (t.tv_sec * 1000) + (t.tv_nsec*1e-6);
}

// Kernel execution
__global__ void nn(float (*a)[n][n], float (*b)[n][n]){

    // Global indices
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Only compute interior points (1 ≤ i,j,k ≤ n-2)
    if (i > 0 && i < n-1 && j > 0 && j < n-1 && k > 0 && k < n-1) {
        a[i][j][k] = 0.75f * (
            b[i-1][j][k] + b[i+1][j][k] +  // x neighbors
            b[i][j-1][k] + b[i][j+1][k] +  // y neighbors
            b[i][j][k-1] + b[i][j][k+1]    // z neighbors
        );
    }

    // Ensure all threads have reached this point before continuation
    __syncthreads();
    

}

int main(int argc, char* argv[]){

    double start, end, total;

    //dim3 dimensions = dim3(n, n, n);
    float a[n][n][n], b[n][n][n];
    for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++){
            for(int k = 0; k < n; k++){
                b[i][j][k] = 2.0f;
                a[i][j][k] = 0.0f;
            }
        }
    }

    // Allocate device memory for 3D arrays
    float (*cuA)[n][n], (*cuB)[n][n];
    hipMalloc(&cuA, sizeof(float)*n*n*n);
    hipMalloc(&cuB, sizeof(float)*n*n*n);
    
    // Copy data to device
    hipMemcpy(cuB, b, sizeof(float)*n*n*n, hipMemcpyHostToDevice);
    hipMemcpy(cuA, a, sizeof(float)*n*n*n, hipMemcpyHostToDevice);

    // Dimensions
    /*
    For 32 x 32 x 32:
    - 6 x 6 x 6 threads per block
    - 8 x 8 x 8 blocks in grid
    */
	int threads = 8;
    dim3 gridDim((n + threads - 1) / threads, (n + threads - 1) / threads, (n + threads - 1) / threads);
    dim3 blockDim(threads, threads, threads);

    // track start of operation
    start = CLOCK();

    // Execute kernel several times
    for(int i = 0; i < 1000; i++){
        nn<<<gridDim, blockDim>>>(cuA, cuB);
    }
    
    hipDeviceSynchronize();
    // track end of operation
    end = CLOCK();

	//cudaDeviceSynchronize();

    // copy memory back to device 3d array
    hipMemcpy(a, cuA, sizeof(float) * n * n * n, hipMemcpyDeviceToHost);

    // Time calculations and printing
    total = end - start;
    std::cout << "Time: " << total << " ms" << std::endl;
    
    // Checking answers
    for(int i = 0; i < 10; i++){
        //std::cout << a[1][1][i] << std::endl;
    }

    hipFree(cuA);
    hipFree(cuB);

    return 0;
}